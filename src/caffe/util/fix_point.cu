#include "hip/hip_runtime.h"
//#include <mutex>
#include <cfloat>

#include "caffe/common.hpp"
#include "caffe/blob.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/fix_point.hpp"

namespace caffe {

template <typename Dtype>
static __global__ void saturate_fix_kernel(const int n, Dtype upper_bound, Dtype lower_bound, Dtype step, const Dtype* x, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
	y[index]=min(max(round(x[index]/step)*step, lower_bound), upper_bound);
  }
}

template <typename Dtype>
void caffe_gpu_fix(const int n, const Dtype*x, Dtype* y, const int bit_level, const int scale) {
	Dtype step=std::pow(Dtype(2), scale-bit_level);
	Dtype lower_bound=-std::pow(Dtype(2), scale);
	Dtype upper_bound=std::pow(Dtype(2), scale)-step;
	saturate_fix_kernel<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
			n, upper_bound, lower_bound, step, x, y);
	CUDA_POST_KERNEL_CHECK;
}
template void caffe_gpu_fix<float>(const int n, const float* x, float* y, const int bit_level, const int scale);
template void caffe_gpu_fix<double>(const int n, const double* x, double* y, const int bit_level, const int scale);

/*
template <typename Dtype>
void caffe_gpu_fix_diff_w(const int n, const Dtype*x, Dtype* y, const int bit_level, const int max_scale, const int min_scale, int& final_scale) {
	final_scale=max_scale;
	Dtype fixed_diff_min = FLT_MAX;
	for(int scale=max_scale; scale>=min_scale; scale--){
		caffe_gpu_fix<Dtype>(n, x, y, bit_level, scale);
		caffe_gpu_sub<Dtype>(n, x, y, y);
		Dtype fixed_diff;
		caffe_gpu_asum(n, y, &fixed_diff);
		if(fixed_diff<fixed_diff_min){
			final_scale=scale;
			fixed_diff_min=fixed_diff;
		}
	}
	caffe_gpu_fix(n, x, y, bit_level, final_scale);
}
template void caffe_gpu_fix_diff_w<float>(const int n, const float* x, float* y, const int bit_level, const int max_scale, const int min_scale, int& final_scale);
template void caffe_gpu_fix_diff_w<double>(const int n, const double* x, double* y, const int bit_level, const int max_scale, const int min_scale, int& final_scale);

template <typename Dtype>
static __global__ void overflow_kernel(const int n, Dtype upper_bound, Dtype lower_bound, const Dtype* x, Dtype* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index]=(x[index]<=upper_bound && x[index]>=lower_bound)?Dtype(0):Dtype(1);
	}
}

template <typename Dtype>
static bool test_overflow(const int n, Dtype upper_bound, Dtype lower_bound, const Dtype* data, Dtype* buffer) {
	overflow_kernel<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n, upper_bound, lower_bound, data, buffer);
	CUDA_POST_KERNEL_CHECK;
	Dtype asum;
	caffe_gpu_asum(n, buffer, &asum);
	return asum>Dtype(0.5);
}

template <typename Dtype>
void caffe_gpu_fix_overflow(const int n, const Dtype* x, Dtype* y, const int bit_level, const int max_scale, const int min_scale, int& final_scale) {
	final_scale=std::max(std::min(final_scale, max_scale), min_scale);
	int search_length=max_scale-min_scale+1;
	if(search_length<2) {
		final_scale=min_scale;
	}
	else {
		Dtype* buffer=y;
		if(x==y) {
			buffer=static_cast<Dtype*>(Caffe::GpuBuffer(n*sizeof(Dtype)));
			CHECK_NOTNULL(buffer);
		}

		vector<Dtype> upper_bound(search_length);
		vector<Dtype> lower_bound(search_length);
		for(int i=0; i<search_length; i++) {
			upper_bound[i]=std::pow(Dtype(2), i+min_scale);
			lower_bound[i]=-upper_bound[i]-std::pow(Dtype(2), i+min_scale-bit_level);
		}

		vector<bool> overflow(search_length);
		vector<bool> tested(search_length, false);

		bool found=false;
		overflow[final_scale-min_scale]=test_overflow(n, upper_bound[final_scale-min_scale],
				lower_bound[final_scale-min_scale], x, buffer);
		tested[final_scale-min_scale]=true;
		if(!overflow[final_scale-min_scale]) {
			if(final_scale==min_scale) {
				found=true;
			}
			else {
				overflow[final_scale-min_scale-1]=test_overflow(n, upper_bound[final_scale-min_scale-1],
						lower_bound[final_scale-min_scale-1], x, buffer);
				tested[final_scale-min_scale-1]=true;
				if(overflow[final_scale-min_scale-1]) {
					found=true;
				}
			}
		}

		if(!found) {
			overflow[0]=true;
			tested[0]=true;
			overflow[search_length-1]=false;
			tested[search_length-1]=true;
			int left=0;
			int right=search_length-1;
			for(;;) {
				int middle=(left+right)/2;
				if(!tested[middle]) {
					overflow[middle]=test_overflow(n, upper_bound[middle], lower_bound[middle], x, buffer);
					tested[middle]=true;
				}
				if(!tested[middle+1]) {
					overflow[middle+1]=test_overflow(n, upper_bound[middle+1], lower_bound[middle+1], x, buffer);
					tested[middle+1]=true;
				}
				if(overflow[middle] && !overflow[middle+1]) {
					final_scale=min_scale+middle+1;
					break;
				}
				else if(!overflow[middle]) {
					right=middle;
				}
				else {
					left=middle+1;
				}
			}
		}
	}
	caffe_gpu_fix(n, x, y, bit_level, final_scale);
}
template void caffe_gpu_fix_overflow<float>(const int n, const float* x, float* y, const int bit_level, const int max_scale, const int min_scale, int& final_scale);
template void caffe_gpu_fix_overflow<double>(const int n, const double* x, double* y, const int bit_level, const int max_scale, const int min_scale, int& final_scale);
*/
}  // namespace caffe
